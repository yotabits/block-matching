#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../hardware_limits.hh"
#include <time.h>


__device__ unsigned int get_index_of(unsigned int x, unsigned int y, unsigned int size_x)
{
	unsigned int index = x + y * size_x;
	return index;
}

__global__ void extract_zone_and_replicate_k(unsigned char *img, unsigned char *output,
										   unsigned int size_x, unsigned int size_y, unsigned int size_x_y_roi,
										   unsigned int y_roi)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//unsigned int actual_pos = tid + starting_index;  //position in the image buffer

	//x y corresponding to thread in the input img
	//x y for tid 0 start at roi starting point
	//unsigned int y_img = actual_pos / size_x;
	//unsigned int x_img = actual_pos - (y_img * size_x);
	unsigned int x_base = tid;
    unsigned int y_base = y_roi;
	if (tid < size_x - size_x_y_roi)
	{
		unsigned long output_index = tid * size_x_y_roi * size_x_y_roi;
		for (unsigned int y = y_base; y < y_base + size_x_y_roi; y++)
		{
			for (unsigned int x = x_base; x < x_base + size_x_y_roi; x++)
			{
				output[output_index] = img[x + y * size_x];
				output_index = output_index + 1;
			}
		}
	}
}

void extract_and_replicate(unsigned char *img_gpu, unsigned int size_x, unsigned int size_y, unsigned int roi_size_x_y,
						   unsigned char *output_gpu_buffer, unsigned int y_roi)
{
	unsigned int blocks_to_compare = size_x - roi_size_x_y;
	unsigned int threads = 0;
	unsigned int blocks = 0;
	get_optimized_thread_blocks(&threads,&blocks, blocks_to_compare);
	//printf("threads %i\n", threads);
	//printf("blocks %i \n", blocks);


	extract_zone_and_replicate_k<<<blocks, threads>>>(img_gpu, output_gpu_buffer, size_x, size_y, roi_size_x_y, y_roi);

}



