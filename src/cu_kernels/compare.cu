#include "hip/hip_runtime.h"

/*
 * compare.cu
 *
 *  Created on: Apr 25, 2017
 *      Author: Thomas Kostas
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "extractor.cuh"
#include "../hardware_limits.hh"

//output, size = nb_blocks in buffer 1 sad index per block
__global__ void sad_k(unsigned char *block_buffer, unsigned int block_size_x_y, unsigned int block_buffer_size,
											 unsigned char *block_reference, unsigned int *output)
{
	__shared__ unsigned int sad[1]; //sad will be in position 0
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int ref_index = tid % (block_size_x_y * block_size_x_y);
	unsigned int output_index = tid / (block_size_x_y* block_size_x_y);
	if (tid < block_buffer_size)
	{
		unsigned int diff = abs(block_buffer[tid] - block_reference[ref_index]);
		block_buffer[tid] = 0;
	    atomicAdd(sad,diff);
	}
	__syncthreads();

	if (threadIdx.x == 0)
	{
		output[output_index] = *sad;
	}
}

 unsigned int *compute_sad(unsigned char *block_buffer, unsigned int block_size_x_y, unsigned int nb_blocks,
		 unsigned char *block_reference, unsigned int *output)
 {
	unsigned int threads = block_size_x_y * block_size_x_y;
	unsigned int blocks;
	unsigned int data_size = block_size_x_y * block_size_x_y * nb_blocks;
	get_blocks(threads, data_size, &blocks);
	//printf("blocks = %i threads %i \n", blocks, threads);
	sad_k<<<blocks, threads>>>(block_buffer, block_size_x_y, data_size, block_reference,output);
	 return output;
 }

